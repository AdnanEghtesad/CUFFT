#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h> 

#define NX 128
#define NY 128
#define NZ 128
#define LX (2*M_PI)
#define LY (2*M_PI)
#define LZ (2*M_PI)

typedef float fftw_complex[2];

#define REAL 0
#define IMAG 1

int main()
{
  double acc_time = 0;
  int acc_n = 0;
	         
  fftw_complex *vx = new fftw_complex[NX*NY*NZ];

  for(int k = 0; k < NZ; k++){
    for(int j = 0; j < NY; j++){
      for(int i = 0; i < NX; i++){
	float x = i * LX/NX;
	vx[(k*NY+j)*NX + i][REAL] = cos(x);
	vx[(k*NY+j)*NX + i][IMAG] = 0.;
      }
    }
  }

  float *d_vx;
  
  struct timespec now, tmstart;
  clock_gettime(CLOCK_REALTIME, &tmstart);
  
  hipMalloc(&d_vx, NX*NY*NZ*sizeof(fftw_complex));
 
  
  hipMemcpy(d_vx, vx, NX*NY*NZ*sizeof(fftw_complex), hipMemcpyHostToDevice);
  
  
  
  hipfftHandle planc2c;
  hipfftPlan3d(&planc2c, NZ,NY, NX, HIPFFT_C2C);
  cufftSetCompatibilityMode(planc2c, CUFFT_COMPATIBILITY_NATIVE);

  
  
  hipfftExecC2C(planc2c, (hipfftComplex *)d_vx, (hipfftComplex *)d_vx, HIPFFT_FORWARD);

  


  hipMemcpy(vx, d_vx, NX*NY*NZ*sizeof(fftw_complex), hipMemcpyDeviceToHost);
  
  clock_gettime(CLOCK_REALTIME, &now);
  acc_time += (now.tv_sec+now.tv_nsec*1e-9) - (tmstart.tv_sec+tmstart.tv_nsec*1e-9);
  acc_n++;
  printf("avg CUFFT time : %g total time %g\n", acc_time / acc_n, acc_time);
 
#if 1									       
  getchar();
#endif

  for(int k = 0; k < NZ; k++){
    for (int j = 0; j < NY; j++){
      for (int i = 0; i < NX; i++){
	printf("(%.3f,%.3f) ",
	       vx[(k*NY + j)*NX + i][REAL]  ,
	       vx[(k*NY + j)*NX + i][IMAG] );
      }
      printf("\n");
    } 
    printf("\n");
  }
  return 0;

}
