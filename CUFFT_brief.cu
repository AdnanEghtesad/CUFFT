#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipfft/hipfft.h>


#define NX 200

#define NY 100

int main(void) {

hipfftHandle plan;

hipfftComplex *data1, *data2;

hipMalloc((void**)&data1, sizeof(hipfftComplex)*NX*NY);

hipMalloc((void**)&data2, sizeof(hipfftComplex)*NX*NY);

/* Create a 2D FFT plan. */

hipfftPlan2d(&plan, NX, NY, CUFFT_DATA_C2C);

/* Use the CUFFT plan to transform the signal out of place.

*/

cufftExecute(plan, data1, data2, HIPFFT_FORWARD);

}
