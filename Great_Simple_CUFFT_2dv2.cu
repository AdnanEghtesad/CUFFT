#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipfft/hipfft.h> 


#define NX 2
#define NY 2
#define LX (2*M_PI)
#define LY (2*M_PI)

#define NUM_POINTS1 1024
#define NUM_POINTS2 1024
#define NUM_POINTS3 1024
#define NUM_POINTS NX*NY //256*256*256 


#define REAL 0
#define IMAG 1


int main(void) {

float *x = new float[NX*NY];
float *y = new float[NX*NY];
float *vx = new float[NX*NY];

 


for(int j = 0; j < NY; j++){
    for(int i = 0; i < NX; i++){
        x[j*NX + i] = i * LX/NX;
        y[j*NX + i] = j * LY/NY;
        vx[j*NX + i] = cos(x[j*NX + i]);
    }
}

/*
    for (int i = 0; i < NUM_POINTS; ++i) {
      float theta = (float)i / (float)(NUM_POINTS) * 2.*M_PI;

#if 0
        vx[i][REAL] = 1.0 * cos(10.0 * theta) +
                          0.5 * cos(25.0 * theta);

        vx[i][IMAG] = 1.0 * sin(10.0 * theta) +
                          0.5 * sin(25.0 * theta);
#else
	vx[i][REAL] = 1.0 * cos(theta);
	vx[i][IMAG] = 0.0;
#endif
    }
*/ 


float *d_vx;
hipMalloc(&d_vx, NX*NY*sizeof(float));
hipMemcpy(d_vx, vx, NX*NY*sizeof(float), hipMemcpyHostToDevice);
hipfftHandle planr2c;
hipfftHandle planc2r;
hipfftPlan2d(&planr2c, NY, NX, HIPFFT_R2C);
hipfftPlan2d(&planc2r, NY, NX, HIPFFT_C2R);
cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE);
cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE);
hipfftExecR2C(planr2c, (hipfftReal *)d_vx, (hipfftComplex *)d_vx);
hipfftExecC2R(planc2r, (hipfftComplex *)d_vx, (hipfftReal *)d_vx);
hipMemcpy(vx, d_vx, NX*NY*sizeof(hipfftReal), hipMemcpyDeviceToHost);
for (int j = 0; j < NY; j++){
    for (int i = 0; i < NX; i++){
        printf("%.3f ", vx[j*NX + i]/(NX*NY));
    }
    printf("\n");
} 

return 0;

}