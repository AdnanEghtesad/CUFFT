#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipfft/hipfft.h> 


#define NX 512
#define NY 512
#define NZ 512
#define LX (2*M_PI)
#define LY (2*M_PI)
#define LZ (2*M_PI)



int main() {
  
 double acc_time;
 int acc_n;
 
float *x = new float[NX*NY*NZ];
float *y = new float[NX*NY*NZ];
float *z = new float[NX*NY*NZ];
float *vx = new float[NX*NY*NZ];

 
 for(int k = 0; k < NZ; k++){
  for(int j = 0; j < NY; j++){
    for(int i = 0; i < NX; i++){
    
        x[j*NX + i] = i * LX/NX;
        y[j*NX + i] = j * LY/NY;
	z[j*NX + i] = k * LZ/NZ;
        vx[j*NX + i] = cos(x[j*NX + i]);
	}
    }
}
float *d_vx;
hipMalloc(&d_vx, NX*NY*NZ*sizeof(hipfftComplex));
hipMemcpy(d_vx, vx, NX*NY*NZ*sizeof(hipfftComplex), hipMemcpyHostToDevice);
hipfftHandle planr2c;
//hipfftHandle planc2r;
hipfftPlan3d(&planr2c, NZ,NY, NX, HIPFFT_R2C);
//hipfftPlan3d(&planc2r, NZ,NY, NX, HIPFFT_C2R);
cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE);
//cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE);

    struct timespec now, tmstart;
    clock_gettime(CLOCK_REALTIME, &tmstart);

hipfftExecR2C(planr2c, (hipfftReal *)d_vx, (hipfftComplex *)d_vx);
//hipfftExecC2R(planc2r, (hipfftComplex *)d_vx, (hipfftReal *)d_vx);
hipMemcpy(vx, d_vx, NX*NY*NZ*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
   
    clock_gettime(CLOCK_REALTIME, &now);
    acc_time += (now.tv_sec+now.tv_nsec*1e-9) - (tmstart.tv_sec+tmstart.tv_nsec*1e-9);
    acc_n++;

      

    printf("avg CUFFT time : %g total time %g\n", acc_time / acc_n, acc_time);
    getchar();


 for(int k = 0; k < NZ; k++){
  for (int j = 0; j < NY; j++){
    for (int i = 0; i < NX; i++){
        printf("%.3f ", vx[j*NX + i]/ (NX*NY*NZ));
    }
    printf("\n");
  } 
 }
return 0;

}